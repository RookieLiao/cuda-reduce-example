#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../utils.h"

extern int cpuReduction(int* idata, size_t size);

// compltely unroll
template<size_t blockSize>
__global__ void reduce5(int* idata_d, int* odata_d, size_t size) {
  extern __shared__ int sdata[];
  size_t tid = threadIdx.x;
  size_t g_idx = (2 * blockDim.x) * blockIdx.x + tid;

  // if out of boundary, just return
  if (g_idx + blockDim.x >= size) { return; }

  sdata[tid] = idata_d[g_idx] + idata_d[g_idx + blockDim.x];
  __syncthreads();

  if (blockSize >= 512) {
    if (tid < 256) { sdata[tid] += sdata[tid + 256]; }
    __syncthreads();
  }

  if (blockSize >= 256) {
    if (tid < 128) { sdata[tid] += sdata[tid + 128]; }
    __syncthreads();
  }

  if (blockSize >= 128) {
    if (tid < 64) { sdata[tid] += sdata[tid + 64]; }
    __syncthreads();
  }

  // unroll last warp
  if (tid < 32) {
    volatile int* temp = static_cast<volatile int*>(sdata);
#pragma unroll
    for (size_t stride = 32; stride > 0; stride >>= 1) { temp[tid] += temp[tid + stride]; }
  }

  if (tid == 0) { odata_d[blockIdx.x] = sdata[0]; }
}

int performCudaReductionV5() {
  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  printf("starting reduction at cuda_v5 ");
  printf("device %d: %s ", dev, deviceProp.name);

  size_t elem_size = 1 << 28;
  printf("    with array size %zu    \n", elem_size);
  size_t bytes = elem_size * sizeof(int);

  // allocate host memory
  int* idata_h = (int*)malloc(bytes);
  int* temp = (int*)malloc(bytes);

  initialize_array(idata_h, elem_size);

  double iStart, iElaps;

  // cpu reduction
  memcpy(temp, idata_h, bytes);

  int cpu_sum = cpuReduction(temp, elem_size);

  // cuda reduce
  constexpr size_t block_size = 128;

  dim3 block(block_size);
  dim3 grid0(((elem_size - 1) / (2 * block_size) + 1));
  dim3 grid1((grid0.x - 1) / (2 * block_size) + 1);
  dim3 grid2((grid1.x - 1) / (2 * block_size) + 1);

  // allocate device memory
  int* idata_d = NULL;
  int* odata_d0 = NULL;
  int* odata_d1 = NULL;
  int* odata_d2 = NULL;

  hipMalloc((void**)(&idata_d), bytes);
  hipMalloc((void**)(&odata_d0), grid0.x * sizeof(int));
  hipMalloc((void**)(&odata_d1), grid1.x * sizeof(int));
  hipMalloc((void**)(&odata_d2), grid2.x * sizeof(int));

  int* odata_h = (int*)malloc(grid2.x * sizeof(int));

  hipMemcpy(idata_d, idata_h, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = seconds();

  size_t smem_size = block_size * sizeof(int);
  reduce5<block_size><<<grid0, block, smem_size>>>(idata_d, odata_d0, elem_size);
  reduce5<block_size><<<grid1, block, smem_size>>>(odata_d0, odata_d1, grid0.x);
  reduce5<block_size><<<grid2, block, smem_size>>>(odata_d1, odata_d2, grid1.x);

  hipMemcpy(odata_h, odata_d2, grid2.x * sizeof(int), hipMemcpyDeviceToHost);
  int gpu_sum = cpuReduction(odata_h, grid2.x);
  iElaps = seconds() - iStart;
  float gpu_bw = bytes / iElaps / 1e9;
  printf("reduction_v5 elapsed %lf ms, bandwidth %lf GB/s\n", iElaps * 1e3, gpu_bw);

  free(idata_h);
  free(temp);

  hipFree(idata_d);
  hipFree(odata_d0);
  hipFree(odata_d1);
  hipFree(odata_d2);

  // check results
  bool bResult = (gpu_sum == cpu_sum);
  if (!bResult) printf("Test failed!\n");
  return EXIT_SUCCESS;
}
